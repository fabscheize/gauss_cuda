#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>

#include "hip/hip_runtime.h"
#include ""
#define EPS 1E-9

__global__ void calculate_matrix(float* matrix, int n, float* det);

int main() {
  for (int n = 3; n < 3004; n += 200) {
    float* matrix = (float*)malloc(n * n * sizeof(float));
    float det = 1.0;

    srand(time(NULL));
    for (int i = 0; i < n * n; i++) matrix[i] = rand() % 10;

    float* cudaMatrix = NULL;
    float* cudadet = NULL;
    hipMalloc(&cudaMatrix, n * n * sizeof(float));
    hipMemcpy(cudaMatrix, matrix, n * n * sizeof(float),
               hipMemcpyHostToDevice);
    hipMalloc(&cudadet, sizeof(float));
    hipMemcpy(cudadet, &det, sizeof(float), hipMemcpyHostToDevice);
    float calculation_time = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    calculate_matrix<<<(n + 900) / 900, 900>>>(cudaMatrix, n, cudadet);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calculation_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(&det, cudadet, sizeof(float), hipMemcpyDeviceToHost);

    printf("Определитель матрицы %d x %d вычислен за %lf сек\n", n, n,
           (float)(calculation_time / 1000));
    free(matrix);
    hipFree(cudaMatrix);
  }
  return 0;
}
__global__ void calculate_matrix(float* matrix, int n, float* det) {
  int xid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = 0; i < n; ++i) {
    int k = i;
    for (int j = i + 1; j < n; ++j) {
      if (fabs(matrix[j * n + i]) > fabs(matrix[k * n + i])) {
        k = j;
      }
    }
    if (fabs(matrix[k * n + i]) < EPS) {
      *det = 0.0;
      break;
    }
    if (i != k) {
      *det = -*det;
      for (int l = 0; l < n; l++) {
        float temp = matrix[i * n + l];
        matrix[i * n + l] = matrix[k * n + l];
        matrix[k * n + l] = temp;
      }
    }
    *det *= matrix[i * n + i];
    for (int j = i + 1; j < n; ++j) {
      if (xid < n) {
        matrix[j * n + i + xid] -=
            matrix[i * n + i + xid] * (matrix[j * n + i] / matrix[i * n + i]);
      }
    }
  }
}
